#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <algorithm>
#include <chrono>
#include <vector>

using namespace std::chrono_literals;
// Kernel definition
__global__ void vectorSum( float const *  v1, float const *  v2, float *  v3)
{
    v3[threadIdx.x] = v1[threadIdx.x] + v2[threadIdx.x];
}


int main()
{
    unsigned int count = 50000000;

    std::vector<float> hVec1(count, 2.2f);
    std::vector<float> hVec2(count, 1.1f);
    std::vector<float> hRes(count, 0.0f);
    std::vector<float> cdRes(count, 0.0f);

    auto st = std::chrono::system_clock::now();

    float* dVec1{};
    hipMalloc(&dVec1, count * sizeof(float));

    float* dVec2{};
    hipMalloc(&dVec2, count * sizeof(float));

    float* dRes{};
    hipMalloc(&dRes, count * sizeof(float));


    hipMemcpy(dVec1, hVec1.data(), count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dVec2, hVec2.data(), count * sizeof(float), hipMemcpyHostToDevice);

    dim3 blocks{};
    dim3 threads{count};
    vectorSum<< < blocks, threads >> > (dVec1, dVec2, dRes);
    hipMemcpy(cdRes.data(), dRes, count*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dVec1);
    hipFree(dVec2);
    hipFree(dRes);

    std::cout << " \n CUDA took " << (std::chrono::system_clock::now() - st).count() << " nano seconds ";


    //To display
    //for (auto& el : hRes) std::cout << el << " ";


    st = std::chrono::system_clock::now();
    std::transform(hVec1.begin(), hVec1.end(), hVec2.begin(), hRes.begin(), [](const auto & i, const auto & j) {return i + j; });
    std::cout << " \n normal took " << (std::chrono::system_clock::now() - st).count() << " nano seconds ";

    int indicate{};
    std::cout<<"\n\n\n\n\n";
    for (size_t i{}; i < cdRes.size(); i++) indicate += cdRes[i] - hRes[i];

    std::cout << "\n\n indicator = " << indicate;
    hipDeviceReset();

}
